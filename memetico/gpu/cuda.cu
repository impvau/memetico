#include "hip/hip_runtime.h"

#include <memetico/gpu/cuda.cuh>

using namespace std;
using namespace cusr;

namespace cusr {
    
    void copySubset(GPUDataset *dataset_struct, vector<size_t> &idxs) {

        // copy label set
        size_t *device_idx_arr;
        hipMalloc((void **) &device_idx_arr, sizeof(size_t) * idxs.size());
        hipMemcpy(device_idx_arr, thrust::raw_pointer_cast(idxs.data()), sizeof(size_t) * idxs.size(), hipMemcpyHostToDevice);
        dataset_struct->subset = device_idx_arr;
        dataset_struct->subset_size = idxs.size();

    }

    void freeSubset(GPUDataset *dataset_struct) {
        hipFree(dataset_struct->subset);
        dataset_struct->subset_size = 0;
    }

    void copyDatasetAndLabel(GPUDataset *dataset_struct, vector<vector<float>> &dataset, vector<float> &label, vector<float> &weight) {
        dataset_struct->dataset_size = dataset.size();
        dataset_struct->is_weighted = weight.size() > 0;

        // format dataset into column-major
        int data_size = dataset.size();
        int variable_num = dataset[0].size();

        vector<float> device_dataset;

        for (int i = 0; i < variable_num; i++) {
            for (int j = 0; j < data_size; j++) {
                device_dataset.emplace_back(dataset[j][i]);
            }
        }

        // copy dataset
        float *device_dataset_arr;
        size_t dataset_pitch;
        auto errr0 = hipMallocPitch((void **) &device_dataset_arr, &dataset_pitch, sizeof(float) * data_size, variable_num);
        auto errr1 = hipMemcpy2D(device_dataset_arr, dataset_pitch, thrust::raw_pointer_cast(device_dataset.data()),
                        sizeof(float) * data_size, sizeof(float) * data_size, variable_num, hipMemcpyHostToDevice);

        // Check dataset
        //float *device_return_arr;
        //auto err1 = hipMemcpy2D(device_return_arr, sizeof(float) * data_size, device_dataset_arr, dataset_pitch, sizeof(float) * data_size, variable_num, hipMemcpyDeviceToHost);
        //for (int i = 0; i < data_size*variable_num; i++) {
        //    cout << device_return_arr[i] << endl;
        //}

        dataset_struct->dataset_pitch = dataset_pitch;
        dataset_struct->dataset = device_dataset_arr;

        // copy label set
        float *device_label_arr;
        auto err0 = hipMalloc((void **) &device_label_arr, sizeof(float) * data_size);
        auto err1 = hipMemcpy(device_label_arr, thrust::raw_pointer_cast(label.data()), sizeof(float) * data_size, hipMemcpyHostToDevice);

        // copy weight set
        float *device_weight_arr;
        auto err2 = hipMalloc((void **) &device_weight_arr, sizeof(float) * data_size);
        auto err3 = hipMemcpy(device_weight_arr, thrust::raw_pointer_cast(weight.data()), sizeof(float) * data_size, hipMemcpyHostToDevice);

        // Check dataset
        //float *device_return_label_arr;
        //auto err2 = hipMemcpy(device_return_label_arr, device_label_arr, sizeof(float)*data_size, hipMemcpyDeviceToHost);
        //for (int i = 0; i < data_size; i++) {
        //    cout << device_return_label_arr[i] << endl;
        //}

        dataset_struct->label = device_label_arr;
        dataset_struct->weight = device_weight_arr;

    }

    void freeDataSetAndLabel(GPUDataset *dataset_struct) {
        hipFree(dataset_struct->dataset);
        hipFree(dataset_struct->label);
    }

    __constant__ float d_nodeValue[MAX_PREFIX_LEN];
    __constant__ float d_nodeType[MAX_PREFIX_LEN];

#define S_OFF THREAD_PER_BLOCK * (CUSR_DEPTH + 1) * blockIdx.x + top * THREAD_PER_BLOCK + threadIdx.x

    __global__ void
    calFitnessGPU(int len, float *ds, int dsPitch, float *label, float *weights, size_t *idxs, float *stack, float *result, float *result_weights,
                        int dataset_size, bool is_subset, bool is_weighted, bool do_print, metric_t metric) {

        extern __shared__ float sharedMem[];

        // Partition sharedMem into two halves
        float* shared = sharedMem;
        float* shared_weights = sharedMem + blockDim.x; // Offset by number of threads in a block

        shared[threadIdx.x] = 0;
        shared_weights[threadIdx.x] = 0;
        
        // each thread is responsible for one datapoint
        int dataset_no = blockIdx.x * THREAD_PER_BLOCK + threadIdx.x;

        if (dataset_no < dataset_size) {

            int top = 0;

            // do stack operation according to the type of each node
            for (int i = len - 1; i >= 0; i--) {

                int node_type = d_nodeType[i];
                float node_value = d_nodeValue[i];

                if (node_type == NodeType::CONST) {
                    stack[S_OFF] = node_value;
                    top++;
                } else if (node_type == NodeType::VAR) {
                    int var_num = node_value;

                    if( is_subset) {
                        stack[S_OFF] = ((float *) ((char *) ds + var_num * dsPitch))[idxs[dataset_no]];
                    } else {
                        stack[S_OFF] = ((float *) ((char *) ds + var_num * dsPitch))[dataset_no];
                    }
                    top++;

                } else if (node_type == NodeType::UFUNC) {
                    int function = node_value;
                    top--;
                    float var1 = stack[S_OFF];
                    if (function == Function::SIN) {
                        stack[S_OFF] = std::sin(var1);
                        top++;
                    } else if (function == Function::COS) {
                        stack[S_OFF] = std::cos(var1);
                        top++;
                    } else if (function == Function::TAN) {
                        stack[S_OFF] = std::tan(var1);
                        top++;
                    } else if (function == Function::LOG) {
                        if (var1 <= 0) {
                            stack[S_OFF] = -1.0f;
                            top++;
                        } else {
                            stack[S_OFF] = std::log(var1);
                            top++;
                        }
                    } else if (function == Function::INV) {
                        if (var1 == 0) {
                            var1 = DELTA;
                        }
                        stack[S_OFF] = 1.0f / var1;
                        top++;
                    }
                } else // if (node_type == NodeType::BFUNC)
                {
                    int function = node_value;
                    top--;
                    float var1 = stack[S_OFF];
                    top--;
                    float var2 = stack[S_OFF];

                    if (function == Function::ADD) {
                        stack[S_OFF] = var1 + var2;
                        top++;
                    } else if (function == Function::SUB) {
                        stack[S_OFF] = var1 - var2;
                        top++;
                    } else if (function == Function::MUL) {
                        stack[S_OFF] = var1 * var2;
                        top++;
                    } else if (function == Function::DIV) {
                        if (var2 == 0) {
                            var2 = DELTA;
                        }
                        stack[S_OFF] = var1 / var2;
                        top++;
                    } else if (function == Function::MAX) {
                        stack[S_OFF] = var1 >= var2 ? var1 : var2;
                        top++;
                    } else if (function == Function::MIN) {
                        stack[S_OFF] = var1 <= var2 ? var1 : var2;
                        top++;
                    }
                }

                if( dataset_no == 1 && do_print)
                    printf("Final ds=1 value %f \n", stack[S_OFF]);
            }

            top--;
            float prefix_value = stack[S_OFF];
            float label_value;
            float weight;
            if(is_subset) {
                label_value = label[idxs[dataset_no]];
                weight = weights[idxs[dataset_no]];
            } else {
                label_value = label[dataset_no];
                weight = weights[dataset_no];
            }
    
            float loss;
            float fitness;
            if(metric == metric_t::mean_square_error || metric == metric_t::root_mean_square_error) {
                loss = prefix_value - label_value;
                fitness = loss * loss;
            }
            if(metric == metric_t::mean_absolute_error) {
                loss = abs(prefix_value - label_value);
                fitness = loss;
            }
            if( is_weighted )
               fitness = fitness*weight;

            shared[threadIdx.x] = fitness;
            shared_weights[threadIdx.x] = weight; // Store the weight

            //if( is_weighted)
            //    printf("Block %d, Thread %d, Fitness: %f, Weight: %f  \n", blockIdx.x, threadIdx.x, shared[threadIdx.x], shared_weights[threadIdx.x] );

        }

        __syncthreads();

        // do parallel reduction
#if THREAD_PER_BLOCK >= 1024
        if (threadIdx.x < 512) {
            shared[threadIdx.x] += shared[threadIdx.x + 512];
            shared_weights[threadIdx.x] += shared_weights[threadIdx.x + 512];
        }
        __syncthreads();
#endif
#if THREAD_PER_BLOCK >= 512
        if (threadIdx.x < 256) {
            shared[threadIdx.x] += shared[threadIdx.x + 256];
            shared_weights[threadIdx.x] += shared_weights[threadIdx.x + 256];
        }
        __syncthreads();
#endif
        if (threadIdx.x < 128) { 
            shared[threadIdx.x] += shared[threadIdx.x + 128];
            shared_weights[threadIdx.x] += shared_weights[threadIdx.x + 128];
        }
        __syncthreads();

        if (threadIdx.x < 64) { 
            shared[threadIdx.x] += shared[threadIdx.x + 64];
            shared_weights[threadIdx.x] += shared_weights[threadIdx.x + 64];
        }
        __syncthreads();

        if (threadIdx.x < 32) { 
            shared[threadIdx.x] += shared[threadIdx.x + 32];
            shared_weights[threadIdx.x] += shared_weights[threadIdx.x + 32];
        }
        __syncthreads();

        if (threadIdx.x < 16) { 
            shared[threadIdx.x] += shared[threadIdx.x + 16];
            shared_weights[threadIdx.x] += shared_weights[threadIdx.x + 16];
        }
        __syncthreads();

        if (threadIdx.x < 8) { 
            shared[threadIdx.x] += shared[threadIdx.x + 8];
            shared_weights[threadIdx.x] += shared_weights[threadIdx.x + 8];
        }
        __syncthreads();

        if (threadIdx.x < 4) { 
            shared[threadIdx.x] += shared[threadIdx.x + 4];
            shared_weights[threadIdx.x] += shared_weights[threadIdx.x + 4];
        }
        __syncthreads();

        if (threadIdx.x < 2) { 
            shared[threadIdx.x] += shared[threadIdx.x + 2];
            shared_weights[threadIdx.x] += shared_weights[threadIdx.x + 2];
        }
        __syncthreads();
        
        if (threadIdx.x < 1) {
            
            shared[threadIdx.x] += shared[threadIdx.x + 1];
            shared_weights[threadIdx.x] += shared_weights[threadIdx.x + 1];

            //if( is_weighted) {
                // Print the reduced values in the first thread of each block
                //printf("Block %d, Reduced Shared: %f, Weight: %f\n", blockIdx.x, shared[0], shared_weights[0]);
            //}

            result[blockIdx.x] = shared[0];
            result_weights[blockIdx.x] = shared_weights[0]; // Sum of weights in this block

        }
    }

    float *mallocStack(int blockNum) {
        float *stack;

        // allocate stack space, the size of which = sizeof(float) * THREAD_PER_BLOCK * (maxDepth + 1)
        hipMalloc((void **) &stack, sizeof(float) * THREAD_PER_BLOCK * (CUSR_DEPTH + 1) * blockNum);

        return stack;
    }

    void calSingleProgram(GPUDataset &dataset, int blockNum, Program &program,
                        float *stack, float *result, float *result_weights, float *h_res, float *h_res_weights, metric_t metric) {

        // --------- restrict the length of prefix ---------
        assert(program.length < MAX_PREFIX_LEN);
        // -------------------------------------------------

        // -------- copy to constant memory --------
        float h_nodeValue[MAX_PREFIX_LEN];
        float h_nodeType[MAX_PREFIX_LEN];

        for (int i = 0; i < program.length; i++) {
            int type = program.prefix[i].node_type;
            h_nodeType[i] = type;
            if (type == NodeType::CONST) {
                h_nodeValue[i] = program.prefix[i].constant;
            } else if (type == NodeType::VAR) {
                h_nodeValue[i] = program.prefix[i].variable;
            } else { // unary function or binary function
                h_nodeValue[i] = program.prefix[i].function;
            }
        }

        hipMemcpyToSymbol(HIP_SYMBOL(d_nodeValue), h_nodeValue, sizeof(float) * program.length);
        hipMemcpyToSymbol(HIP_SYMBOL(d_nodeType), h_nodeType, sizeof(float) * program.length);

        int size;
        if( dataset.subset_size > 0 )
            size = dataset.subset_size;
        else 
            size = dataset.dataset_size;

        // -------- calculation and synchronization --------

        //calFitnessGPU<<<blockNum, THREAD_PER_BLOCK, sizeof(float) * THREAD_PER_BLOCK>>>
                    //(program.length, dataset.dataset, dataset.dataset_pitch, dataset.label, dataset.weight, dataset.subset, stack, result, result_weights,
                        //size, dataset.subset_size > 0, dataset.is_weighted, true, metric);
        
        calFitnessGPU<<<blockNum, THREAD_PER_BLOCK, sizeof(float) * THREAD_PER_BLOCK * 2>>>
            (program.length, dataset.dataset, dataset.dataset_pitch, dataset.label, dataset.weight, dataset.subset, stack, result, result_weights,
            size, dataset.subset_size > 0, dataset.is_weighted, false, metric);

        

        hipDeviceSynchronize();

        // -------- reduction on the result --------
        hipMemcpy(h_res, result, sizeof(float) * blockNum, hipMemcpyDeviceToHost);
        hipMemcpy(h_res_weights, result_weights, sizeof(float) * blockNum, hipMemcpyDeviceToHost); // Get weights

        float total_fitness = 0;
        float total_weights = 0;

        for (int i = 0; i < blockNum; i++) {
            total_fitness += h_res[i];
            total_weights += h_res_weights[i]; // Sum up weights
        }

        // Calculate final weighted fitness
        if (metric == metric_t::mean_absolute_error || metric == metric_t::mean_square_error) {
            program.fitness = dataset.is_weighted ? total_fitness / total_weights : total_fitness / (float) size;
        } else if (metric == metric_t::root_mean_square_error) {
            program.fitness = dataset.is_weighted > 0 ? sqrt(total_fitness / total_weights) : sqrt(total_fitness / (float) size);
        }
        
        total_fitness = 0;
        total_weights = 0;

    }

    float 
    calculateFitness(GPUDataset &dataset, int blockNum, vector<Program> &population, metric_t metric) {

        // allocate space for result
        float *result;
        float *result_weights;
        hipMalloc((void **) &result, sizeof(float) * blockNum);
        hipMalloc((void **) &result_weights, sizeof(float) * blockNum);

        // allocate stack space
        float *stack = mallocStack(blockNum);

        // save result and do CPU side reduction
        float *h_res = new float[blockNum];
        float *h_res_weights = new float[blockNum];

        // evaluate fitness for each program in the population
        calSingleProgram(dataset, blockNum, population[0], stack, result, result_weights, h_res, h_res_weights, metric);

        // free memory space
        hipFree(result);
        hipFree(result_weights);
        hipFree(stack);
        delete[] h_res;
        delete[] h_res_weights;

        return population[0].fitness;
    }

}

